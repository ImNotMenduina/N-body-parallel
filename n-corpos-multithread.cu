
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <time.h>
#include <math.h>
#include <string.h>
#include <omp.h>
#include "hip/hip_runtime.h"


#define MASSA 1
#define EPSILON 1E-9
#define ALING 1024

typedef struct vetor
{
    double x, y, z;
} VETOR;

typedef struct posicao
{
    double x, y, z;
} POSICAO;

typedef struct particula
{
    POSICAO coord;
    VETOR forca_sofrida;
    VETOR velocidade;
} PARTICULA;

void inicializador(PARTICULA *particula, int quantidade)
{
    srand(42);
    memset(particula, 0x00, quantidade * sizeof(PARTICULA));
    for (int i = 0; i < quantidade ; i++){
        particula[i].coord.x =  2.0 * (rand() / (double)RAND_MAX) - 1.0;
        particula[i].coord.y =  2.0 * (rand() / (double)RAND_MAX) - 1.0;
        particula[i].coord.z =  2.0 * (rand() / (double)RAND_MAX) - 1.0;
      }
}

void printLog(PARTICULA *particles, int quantParticulas, int timestep, char* type)
{
    char path[100] = "../";
    sprintf(path, "Log/%s/Log%d-log.txt", type, timestep);
    fprintf(stdout, "Saving file [%s] ", path); fflush(stdout);
    FILE *arquivo = fopen(path, "w+");
    for(int i = 0; i < quantParticulas; i++){
        fprintf(arquivo, "%d \t %.10f %.10f %.10f \t %.10f %.10f %.10f \t %.10f %.10f %.10f \n",
         i,  
            particles[i].coord.x, particles[i].coord.y, particles[i].coord.z,

            particles[i].velocidade.x, particles[i].velocidade.y, particles[i].velocidade.z, 
            
            particles[i].forca_sofrida.x, particles[i].forca_sofrida.y, particles[i].forca_sofrida.z);
    }
    fclose(arquivo);
    fprintf(stdout, "[OK]\n"); fflush(stdout);
}

__global__ void simulacao(PARTICULA* particula, int quantParticulas, int timesteps, double dt);

// __device__ void calcula_forca(PARTICULA* particula, int quantParticulas, double dt) {

// }

__global__ void simulacao(PARTICULA* particula, int quantParticulas, double dt)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < quantParticulas) {

        for(int j = 0; j < quantParticulas; j++) {

            if (j != tid) {
                double dx, dy, dz;

                dx = 0.0f, dy = 0.0f, dz = 0.0f;

                dx = particula[j].coord.x - particula[tid].coord.x; // influencia da força no 
                dy = particula[j].coord.y - particula[tid].coord.y; // vetor decomposto, em cada eixo
                dz = particula[j].coord.z - particula[tid].coord.z;

                double distancia = sqrt(pow(dx, 2) + pow(dy, 2) + pow(dz, 2) + EPSILON);

                double forca = 1.0 / pow(distancia, 2); //para nao dar zero

                particula[j].forca_sofrida.x += dx * forca;
                particula[j].forca_sofrida.y += dy * forca;
                particula[j].forca_sofrida.z += dz * forca;

               
            }

            particula[tid].velocidade.x += dt *  particula[tid].forca_sofrida.x;
            particula[tid].velocidade.y += dt *  particula[tid].forca_sofrida.y;
            particula[tid].velocidade.z += dt *  particula[tid].forca_sofrida.z;

            particula[tid].coord.x += dt *  particula[tid].velocidade.x;
            particula[tid].coord.y += dt *  particula[tid].velocidade.y;
            particula[tid].coord.z += dt *  particula[tid].velocidade.z;             
        }
    }
}


int main (int ac, char **av)
{
    int timesteps = atoi(av[1]), quantParticulas = atoi(av[2]), flagSave = atoi(av[3]);

    clock_t t;
    t = clock();

    char logFile[1024];
    double       dt        =  0.01f;
    PARTICULA *particulas = NULL;
    PARTICULA* d_particula;

    strcpy(logFile, av[4]);

    fprintf(stdout, "\nSistema de particulas P2P(particula a particula)\n");
    fprintf(stdout, "Memória utilizada %lu bytes \n", quantParticulas * sizeof(PARTICULA));
    fprintf(stdout, "Arquivo %s \n", logFile);

    particulas = (PARTICULA *) aligned_alloc(ALING, quantParticulas * sizeof(PARTICULA));
    assert(particulas != NULL);

    inicializador(particulas, quantParticulas);

    int block_size = 8;
    int grid_size = ((quantParticulas + block_size - 1) / block_size);

    hipMalloc((void**)&d_particula, sizeof(PARTICULA) * quantParticulas);

    for (int i=0; i < timesteps; i++) {
        hipMalloc((void**)&d_particula, sizeof(PARTICULA) * quantParticulas);

        hipMemcpy(d_particula, particulas, sizeof(PARTICULA) * quantParticulas, hipMemcpyHostToDevice);

        simulacao<<<grid_size,block_size>>>(d_particula, quantParticulas, dt);

        hipMemcpy(particulas, d_particula, sizeof(PARTICULA) * quantParticulas, hipMemcpyDeviceToHost);
        
        hipFree(d_particula);
    }
    
    t = clock() - t;
    double time_taken = ((double)t)/CLOCKS_PER_SEC;
    fprintf(stdout, "Tempo gasto: %lf (s) \n\n", time_taken);

    FILE *log = fopen(logFile, "a+");
    assert(log != NULL);
    fprintf(log, "Timesteps: %d\nNúmero de Particulas: %d\nMemória em bytes:%lu\nTempo em segundos:%lf\n-----------------------------\n",timesteps,quantParticulas,quantParticulas * sizeof(particulas), time_taken);
    fclose(log);

    if (flagSave == 1)
          printLog(particulas, quantParticulas, timesteps, "ParallelCuda");
    free(particulas);
}

